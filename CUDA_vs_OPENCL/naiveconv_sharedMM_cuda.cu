//**********************************************************************
//                                                                     *
//               University Of North Carolina Charlotte                *
//                                                                     *
//Program: Convolution                                                 *
//Description: This program is to do convolution calculation           *
//             - CUDA                                                  *
//             - Direct convolution with shared memory                 *
//                                                                     *
//File Name: naivecon.c , naiveconv_kernel.cl                          *
//File Version: 1.0                                                    *
//Baseline: Homework_2                                                 *
//                                                                     *
//Course: ECGR 6090 Heterogeneous Computing                            *
//                                                                     *
//Programmed by: Yu Liu                                                * 
//Under Suppervision of: Dr. Hamed Tabkhi                              *
//                                                                     *
//Input file: images/viptraffic0.ppm ...  images/viptraffic119.ppm     *
//Output file: none                                                    *
//**********************************************************************  
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define BLOCKSIZE 256
#define HEIGHT 160
#define WIDTH 120
#define FLTSIZE 3 //filter size
#define PADDING 0
#define STRIDE 1

//**********************************************************************
// Function Name: convolution (Kernel)                                 *
// Description: - Execute direct(naive) convolution                    *
//              - CUDA_global memory                                   *
// Input file: none                                                    *
// Output file: none                                                   *
// Return: none                                                        *
//**********************************************************************
__global__ void convolution(unsigned char *image_d, unsigned char *output_d, float* flt, int convWidth, int convHeight, int filterSize)
{
	int i, j, col, row;
	int r, g, b;

	col = blockIdx.x * blockDim.x + threadIdx.x; //image width *3
	row = blockIdx.y * blockDim.y + threadIdx.y;

	__shared__ float filter[FLTSIZE*FLTSIZE];
	for (i = 0; i < filterSize; i++)
	{
		for (j = 0; j < filterSize; j++)
		{
			filter[i*filterSize + j] = flt[i*filterSize + j];
		}
	}

	__syncthreads();

	if (col < (WIDTH - PADDING + 1)*3 && row < (HEIGHT - PADDING + 1))
	{
		r = 0;
		g = 0;
		b = 0;

		for (i = 0; i < filterSize; i++)
		{
			for (j = 0; j < filterSize; j++)
			{
				r += filter[i*filterSize + j] * image_d[3 * (row + i)*WIDTH + col + j]; //R
				g += filter[i*filterSize + j] * image_d[3 * (row + i)*WIDTH + col + j + 1];//G
				b += filter[i*filterSize + j] * image_d[3 * (row + i)*WIDTH + col + j + 2]; //B
			}

		}

		output_d[row * convWidth + col] = r;
		output_d[row * convWidth + col + 1] = g;
		output_d[row * convWidth + col + 2] = b;

	}
}

//**********************************************************************
// Function Name: decode_image                                         *
// Description: - read image in ppm formate, read the data of array    *
//                named frame[]                                        *
// Input file: image file : viptrafficX.ppm                            *
// Output file: none                                                   *
// Return: 0 if success                                                *
//**********************************************************************

int decode_image(unsigned char frame[HEIGHT * WIDTH * 3], char filename[])
{
	FILE *pFile;
	pFile = fopen(filename, "r");
	fseek(pFile, 15L, SEEK_SET);//In ppm file, the first 15 bytes are content of "p6,120 160, 255", image data is from 16th bytes

	fread(frame, sizeof(unsigned char), HEIGHT * WIDTH * 3 + 15, pFile);
	fclose(pFile);
	return 0;
}

//**********************************************************************
// Function Name:randomInit                                            *
// Description: - Generate random value to an float array              *
//                                                                     *
// Input file: none                                                    *
// Output file: none                                                   *
// Return: kernel file size                                            *
//**********************************************************************
int randomInit(float* data, int size, int range) // random form 0/255 to 255/255
{
	int i;
	srand(time(NULL));
	for (i = 0; i < size; i++)
	{
		data[i] = rand() % range / (float)range;
	}
	//for (i = 0; i < size; i++) printf("%f;", data[i]); // for debugging
	return 0;
}

//**********************************************************************
// Function Name:Main                                                  *
// Description: - Main function on host, configure the kernel parameter*
//                and run kernel                                       *
// Input file: none                                                    *
// Output file: none                                                   *
// Return: 0 if success                                                *
//**********************************************************************
int main(void)
{
	int fltsz = FLTSIZE;
	int convWidth = (WIDTH - FLTSIZE + 2 * PADDING) / STRIDE + 1;  //convolution width with padding
	int convHeight = (HEIGHT - FLTSIZE + 2 * PADDING) / STRIDE + 1;  //convolution width with padding
	int imagecount = 0; //counter for 120 images	
	unsigned char *image_d, *output_d;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float kernelExecTimeNs = 0;
	float timer;

	float* filter = (float*)malloc(FLTSIZE*FLTSIZE * sizeof(float));
	unsigned char* image = (unsigned char*)malloc(HEIGHT * WIDTH * sizeof(unsigned char) * 3);
	unsigned char* output = (unsigned char*)malloc(convHeight * convWidth * 3 * sizeof(unsigned char));
	randomInit(filter, FLTSIZE*FLTSIZE, 255); //initialize filter

	hipMalloc((void**)&image_d, HEIGHT*WIDTH * sizeof(unsigned char) * 3);
	hipMalloc((void**)&output_d, convHeight * convWidth * 3 * sizeof(unsigned char));

	while (imagecount < 120)
	{
		char filename[50];//file length upto 50
		sprintf(filename, "images/viptraffic%d.ppm", imagecount);//read viptrafficX.ppm
		decode_image(image, filename); //get image data from file
		imagecount++;


		//Copy from host to device
		hipMemcpy(image_d, image, HEIGHT*WIDTH * sizeof(unsigned char) * 3, hipMemcpyHostToDevice);

		dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
		dim3 dimGrid((WIDTH * 3 + BLOCKSIZE - 1) / BLOCKSIZE, (HEIGHT + BLOCKSIZE - 1) / BLOCKSIZE);

		hipEventRecord(start, 0);
		convolution <<<dimGrid, dimBlock >>> (image_d, output_d, filter, convWidth, convHeight, fltsz);//Block-thread

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		//Copy from device to host
		hipMemcpy(output, output_d, convHeight * convWidth * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

		hipEventElapsedTime(&timer, start, stop);
		kernelExecTimeNs += timer;
	}

	//Free memory allocation
	hipFree(output_d);
	hipFree(image_d);
	free(output);
	free(image);

	printf("Cumputing done!  Shared memory applied in CUDA.\n");
	printf("Image amount:%d;  Image size:%d x %d;  Padding:%d;  Stride:%d;  Filter Size:%d.\n", imagecount, WIDTH, HEIGHT, PADDING, STRIDE, FLTSIZE);
	printf("Kernel Execution time: %f milli seconds\n", kernelExecTimeNs);
	//system("pause");

	return EXIT_SUCCESS;
}
