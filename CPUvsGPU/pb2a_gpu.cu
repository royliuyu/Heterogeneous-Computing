//**********************************************************************
//                                                                     *
//               University Of North Carolina Charlotte                *
//                                                                     *
//Program: Vecotr adder                                                *
//Description: This program is for testing GPU performance with one    *
//             stencil.                                                *
//                                                                     *
//                                                                     *
//File Name: pb2a_gpu.cu                                                 *
//File Version: 1.0                                                    *
//Baseline: Homework_0                                                 *
//                                                                     *
//Course: ECGR6090- Heterogeneous Computing                            *
//                                                                     *
//Programmed by: Roy Liu                                               * 
//Under Suppervision of: Dr. Hamed Tabkhi                              *
//                                                                     *
//Input file: No                                                       *
//                                                                     *
//Output:Time of program running                                       *
//**********************************************************************   
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#include<hip/hip_runtime.h>  

#define N          10000
#define RADIUS     8
#define BLOCK_SIZE 128

void random_ints(int *r, int n);

__global__ void stencil_1d(int *in, int *out)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	
	int j;
		int result=0;
		for (j=0; j<(1+2*RADIUS);j++)
		{
			result += in[i];
			in+=j;
		}
		
 	out[i]=result;
}

int main()
{
	//for counting run time
    struct timeval start, end;
    float timer;
    gettimeofday(&start, NULL);
    
	int*in, *d_in, *out, *d_out;
   	int n;
	n = N;
   
    int size = (n+2*RADIUS)*sizeof(int);

    // data initializing
    in = (int *)malloc(size); random_ints(in, n);
    out = (int *)malloc(size); 


    //for (int i=0;i<n;i++)	printf("%d\n",a[i]);//for testing
    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out, size);


   
   // Define kernel,block:(1024*1024/512)��512 threds each block
    dim3 dimGrid(n/BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE); //each block has X threads

    // kernel
    stencil_1d<<<dimGrid, dimBlock>>>(d_in, d_out);
	


	// cleanup
	free(in);
	free(out);
	hipFree(d_in);
	hipFree(d_out);
	
 	gettimeofday(&end, NULL);
    timer = 1000000 * (end.tv_sec - start.tv_sec) + end.tv_usec - start.tv_usec;
    printf("Data number is: %d\nBlocksize is: %d\nRadius is: %d\nRunning time is: %f ms\n", n,BLOCK_SIZE,RADIUS,timer/1000);
     
    return 0;
}

//**********************************************************************
// Function Name: random_ints                                          *
// Description: - Generate random integer                              *
// Input : None                                                        *
// Output : Random integer                                             *
// Return: None                                                        *
//**********************************************************************
void random_ints(int* r, int n)
{
	int i;
 	for (i=0; i < n+2*RADIUS; ++i)
 	{
 	 	r[i] = rand()/2;
 	 	
 	}

}
